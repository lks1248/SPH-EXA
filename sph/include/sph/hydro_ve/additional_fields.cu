#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2023 CSCS, ETH Zurich
 *               2023 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief additional fields GPU i-loop driver
 *
 * @author Lukas Schmidt
 */

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"

#include "sph/sph_gpu.hpp"
#include "sph/particles_data.hpp"
#include "sph/hydro_ve/additional_fields_kern.hpp"

namespace sph
{
namespace cuda
{

using cstone::GpuConfig;
using cstone::LocalIndex;
using cstone::TravConfig;
using cstone::TreeNodeIndex;

template<class T, class Tm, class Tc, class KeyType>
__global__ void markRampGPU(unsigned ngmax, const cstone::Box<Tc> box, const LocalIndex* grpStart,
                            const LocalIndex* grpEnd, LocalIndex numGroups,
                            const cstone::OctreeNsView<Tc, KeyType> tree, const Tc* x, const Tc* y, const Tc* z,
                            const T* h, const T* kx, const T* xm, const Tm* m, T* markRamp, T Atmin, T Atmax, T ramp,
                            LocalIndex* nidx, TreeNodeIndex* globalPool)
{
    unsigned laneIdx     = threadIdx.x & (GpuConfig::warpSize - 1);
    unsigned targetIdx   = 0;
    unsigned warpIdxGrid = (blockDim.x * blockIdx.x + threadIdx.x) >> GpuConfig::warpSizeLog2;

    cstone::LocalIndex* neighborsWarp = nidx + ngmax * TravConfig::targetSize * warpIdxGrid;

    while (true)
    {
        // first thread in warp grabs next target
        if (laneIdx == 0) { targetIdx = atomicAdd(&cstone::targetCounterGlob, 1); }
        targetIdx = cstone::shflSync(targetIdx, 0);

        if (targetIdx >= numGroups) return;

        cstone::LocalIndex bodyBegin = grpStart[targetIdx];
        cstone::LocalIndex bodyEnd   = grpEnd[targetIdx];
        cstone::LocalIndex i         = bodyBegin + laneIdx;

        auto ncTrue = traverseNeighbors(bodyBegin, bodyEnd, x, y, z, h, tree, box, neighborsWarp, ngmax, globalPool);

        if (i >= last) continue;

        unsigned ncCapped = stl::min(ncTrue[0], ngmax);
        markRampJLoop<TravConfig::targetSize>(i, neighborsWarp + laneIdx, ncCapped, Atmin, Atmax, ramp, kx, xm, m,
                                              markRamp);
    }
}

template<class Dataset>
void computeMarkRamp(const GroupView& grp, Dataset& d, const cstone::Box<typename Dataset::RealType>& box)
{

    auto [traversalPool, nidxPool] = cstone::allocateNcStacks(d.devData.traversalStack, d.ngmax);
    cstone::resetTraversalCounters<<<1, 1>>>();

    markRampGPU<<<TravConfig::numBlocks(), TravConfig::numThreads>>>(
        d.ngmax, box, grp.groupStart, grp.groupEnd, grp.numGroups, d.treeView, rawPtr(d.devData.x), rawPtr(d.devData.y),
        rawPtr(d.devData.z), rawPtr(d.devData.h), rawPtr(d.devData.kx), rawPtr(d.devData.xm), rawPtr(d.devData.m),
        rawPtr(d.devData.markRamp), d.Atmin, d.Atmax, d.ramp, nidxPool, traversalPool);
    checkGpuErrors(hipDeviceSynchronize());
}

template void computeMarkRamp(const GroupView& grp, sphexa::ParticlesData<cstone::GpuTag>& d,
                              const cstone::Box<SphTypes::CoordinateType>&);

} // namespace cuda
} // namespace sph
