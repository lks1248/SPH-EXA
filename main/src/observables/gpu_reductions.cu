#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2022 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief reductions for different observables on the GPU
 *
 * @author Lukas Schmidt
 */

#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "cstone/util/tuple.hpp"
#include "gpu_reductions.h"
#include "sph/positions.hpp"

namespace sphexa
{

using cstone::Box;
using cstone::Vec3;
using thrust::get;

//!@brief functor for Kelvin-Helmholtz growth rate
template<class Tc, class Tv, class Tm>
struct GrowthRate
{
    HOST_DEVICE_FUN
    thrust::tuple<double, double, double> operator()(const thrust::tuple<Tc, Tc, Tv, Tm, Tm>& p)
    {
        Tc xi   = get<0>(p);
        Tc yi   = get<1>(p);
        Tv vyi  = get<2>(p);
        Tm xmi  = get<3>(p);
        Tm kxi  = get<4>(p);
        Tc voli = xmi / kxi;
        Tc aux;

        if (yi < ybox * Tc(0.5)) { aux = std::exp(-4.0 * M_PI * std::abs(yi - 0.25)); }
        else { aux = std::exp(-4.0 * M_PI * std::abs(ybox - yi - 0.25)); }
        Tc si = vyi * voli * std::sin(4.0 * M_PI * xi) * aux;
        Tc ci = vyi * voli * std::cos(4.0 * M_PI * xi) * aux;
        Tc di = voli * aux;

        return thrust::make_tuple(si, ci, di);
    }

    Tc ybox;
};

template<class Tc, class Tv, class Tm>
std::tuple<double, double, double> gpuGrowthRate(const Tc* x, const Tc* y, const Tv* vy, const Tm* xm, const Tm* kx,
                                                 const cstone::Box<Tc>& box, size_t startIndex, size_t endIndex)
{
    auto it1 = thrust::make_zip_iterator(
        thrust::make_tuple(x + startIndex, y + startIndex, vy + startIndex, xm + startIndex, kx + startIndex));

    auto it2 = thrust::make_zip_iterator(
        thrust::make_tuple(x + endIndex, y + endIndex, vy + endIndex, xm + endIndex, kx + endIndex));

    auto plus = util::TuplePlus<thrust::tuple<double, double, double>>{};
    auto init = thrust::make_tuple(0.0, 0.0, 0.0);

    auto result = thrust::transform_reduce(thrust::device, it1, it2, GrowthRate<Tc, Tm, Tv>{box.ly()}, init, plus);

    return {get<0>(result), get<1>(result), get<2>(result)};
}

#define GROWTH_GPU(Tc, Tv, Tm)                                                                                         \
    template std::tuple<double, double, double> gpuGrowthRate(const Tc* x, const Tc* y, const Tv* vy, const Tm* xm,    \
                                                              const Tm* kx, const cstone::Box<Tc>& box,                \
                                                              size_t startIndex, size_t endIndex)

GROWTH_GPU(double, double, double);
GROWTH_GPU(double, double, float);
GROWTH_GPU(double, float, float);
GROWTH_GPU(float, float, float);

//!@brief functor for the machSquare summing
template<class Tc, class Tv>
struct MachSquareSum
{
    HOST_DEVICE_FUN
    double operator()(const thrust::tuple<Tv, Tv, Tv, Tc>& p)
    {
        Vec3<double> V{get<0>(p), get<1>(p), get<2>(p)};
        Tc           c = get<3>(p);
        return norm2(V) / (double(c) * c);
    }
};

template<class Tc, class Tv>
double machSquareSumGpu(const Tv* vx, const Tv* vy, const Tv* vz, const Tc* c, size_t first, size_t last)
{
    auto it1 = thrust::make_zip_iterator(thrust::make_tuple(vx + first, vy + first, vz + first, c + first));
    auto it2 = thrust::make_zip_iterator(thrust::make_tuple(vx + last, vy + last, vz + last, c + last));

    auto   plus               = thrust::plus<double>{};
    double localMachSquareSum = 0.0;

    localMachSquareSum =
        thrust::transform_reduce(thrust::device, it1, it2, MachSquareSum<Tc, Tv>{}, localMachSquareSum, plus);

    return localMachSquareSum;
}

#define MACH_GPU(Tc, Tv)                                                                                               \
    template double machSquareSumGpu(const Tv* vx, const Tv* vy, const Tv* vz, const Tc* c, size_t, size_t)

MACH_GPU(double, double);
MACH_GPU(double, float);
MACH_GPU(float, float);

//!@brief functor to count particles that belong to the cloud
template<class Tc, class Tt, class Tm>
struct Survivors
{
    HOST_DEVICE_FUN
    double operator()(const thrust::tuple<Tt, Tc, Tc, Tm>& p)
    {
        size_t isCloud;
        Tt     temp = get<0>(p);
        Tc     kx   = get<1>(p);
        Tc     xm   = get<2>(p);
        Tm     m    = get<3>(p);
        Tc     rhoi = kx / xm * m;
        if (rhoi >= 0.64 * rhoBubble && temp <= 0.9 * tempWind) { isCloud = 1; }
        else { isCloud = 0; }
        return isCloud;
    }

    double rhoBubble;
    double tempWind;
};
template<class Tc, class Tt, class Tm>
size_t survivorsGpu(const Tt* temp, const Tt* kx, const Tc* xmass, const Tm* m, double rhoBubble, double tempWind,
                    size_t first, size_t last)
{
    auto it1 = thrust::make_zip_iterator(thrust::make_tuple(temp + first, kx + first, xmass + first, m + first));
    auto it2 = thrust::make_zip_iterator(thrust::make_tuple(temp + last, kx + last, xmass + last, m + last));

    auto plus = thrust::plus<size_t>{};

    size_t localSurvivors = 0;

    localSurvivors = thrust::transform_reduce(thrust::device, it1, it2, Survivors<Tc, Tt, Tm>{rhoBubble, tempWind},
                                              localSurvivors, plus);

    return localSurvivors;
}

#define SURVIVORS(Tc, Tt, Tm)                                                                                          \
    template size_t survivorsGpu(const Tt* temp, const Tt* kx, const Tc* xmass, const Tm* m, double, double, size_t,   \
                                 size_t)

SURVIVORS(double, double, double);
SURVIVORS(double, double, float);
SURVIVORS(double, float, float);
SURVIVORS(float, float, float);

template<class T, class Tc, class Tm>
struct MarkRampCond
{
    HOST_DEVICE_FUN
    void operator()(thrust::tuple<T, T, T, Tm, AuxT<T>, AuxT<T>> p)
    {
        T  h        = get<0>(p);
        T  y        = get<1>(p);
        T  vy       = get<2>(p);
        Tm markRamp = get<3>(p);
        if (markRamp > 0.05 && !sph::fbcCheck(y, h, ymax, ymin, (bool)true))
        {
            thrust::get<4>(p) = {y, vy};
            thrust::get<5>(p) = {y, vy};
        }
    }
    Tc ymin;
    Tc ymax;
};

template<class T, class Tc, class Tm>
std::tuple<std::vector<AuxT<T>>, std::vector<AuxT<T>>> localGrowthRateRTGpu(size_t first, size_t last, Tc ymin, Tc ymax,
                                                                            const T* h, const T* y, const T* vy,
                                                                            const Tm* markRamp)
{
    thrust::device_vector<AuxT<T>> targetUp(last - first);
    thrust::device_vector<AuxT<T>> targetDown(last - first);

    auto it1 = thrust::make_zip_iterator(
        thrust::make_tuple(h + first, y + first, vy + first, markRamp + first, targetUp.begin(), targetDown.begin()));
    auto it2 = thrust::make_zip_iterator(
        thrust::make_tuple(h + last, y + last, vy + last, markRamp + last, targetUp.end(), targetDown.end()));

    thrust::for_each(thrust::device, it1, it2, MarkRampCond<T, Tc, Tm>{ymin, ymax});

    thrust::sort(thrust::device, targetUp.begin(), targetUp.end(), greaterRT());
    thrust::sort(thrust::device, targetDown.begin(), targetDown.end(), lowerRT());

    targetUp.resize(50);
    targetDown.resize(50);

    std::vector<AuxT<T>> retUp(50);
    std::vector<AuxT<T>> retDown(50);
    thrust::copy(targetUp.begin(), targetUp.end(), retUp.begin());
    thrust::copy(targetDown.begin(), targetDown.end(), retDown.begin());

    return std::make_tuple(retUp, retDown);
}

#define RTGROWTH(T, Tc, Tm)                                                                                            \
    template std::tuple<std::vector<AuxT<T>>, std::vector<AuxT<T>>> localGrowthRateRTGpu(                              \
        size_t first, size_t last, Tc ymin, Tc ymax, const T* h, const T* y, const T* vy, const Tm* markRamp);

RTGROWTH(double, double, double);
RTGROWTH(double, float, double);
RTGROWTH(double, double, float);
RTGROWTH(double, float, float);
RTGROWTH(float, float, float);

} // namespace sphexa
