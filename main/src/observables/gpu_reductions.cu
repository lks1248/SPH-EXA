#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2022 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief reductions for different observables on the GPU
 *
 * @author Lukas Schmidt
 */

#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/remove.h>

#include "cstone/util/tuple.hpp"
#include "gpu_reductions.h"
#include "sph/positions.hpp"

namespace sphexa
{

using cstone::Box;
using cstone::Vec3;
using thrust::get;

//!@brief functor for Kelvin-Helmholtz growth rate
template<class Tc, class Tv, class T>
struct GrowthRate
{
    HOST_DEVICE_FUN
    thrust::tuple<double, double, double> operator()(const thrust::tuple<Tc, Tc, Tv, T, T>& p)
    {
        auto [xi, yi, vyi, xmi, kxi] = p;
        auto voli                    = xmi / kxi;
        Tc   aux;

        if (yi < ybox * Tc(0.5)) { aux = std::exp(-4.0 * M_PI * std::abs(yi - 0.25)); }
        else { aux = std::exp(-4.0 * M_PI * std::abs(ybox - yi - 0.25)); }
        Tc si = vyi * voli * std::sin(4.0 * M_PI * xi) * aux;
        Tc ci = vyi * voli * std::cos(4.0 * M_PI * xi) * aux;
        Tc di = voli * aux;

        return thrust::make_tuple(si, ci, di);
    }

    Tc ybox;
};

template<class Tc, class Tv, class T>
std::tuple<double, double, double> gpuGrowthRate(const Tc* x, const Tc* y, const Tv* vy, const T* xm, const T* kx,
                                                 const cstone::Box<Tc>& box, size_t startIndex, size_t endIndex)
{
    auto it1 = thrust::make_zip_iterator(
        thrust::make_tuple(x + startIndex, y + startIndex, vy + startIndex, xm + startIndex, kx + startIndex));

    auto it2 = thrust::make_zip_iterator(
        thrust::make_tuple(x + endIndex, y + endIndex, vy + endIndex, xm + endIndex, kx + endIndex));

    auto plus = util::TuplePlus<thrust::tuple<double, double, double>>{};
    auto init = thrust::make_tuple(0.0, 0.0, 0.0);

    auto result = thrust::transform_reduce(thrust::device, it1, it2, GrowthRate<Tc, T, Tv>{box.ly()}, init, plus);

    return {get<0>(result), get<1>(result), get<2>(result)};
}

#define GROWTH_GPU(Tc, Tv, T)                                                                                          \
    template std::tuple<double, double, double> gpuGrowthRate(const Tc* x, const Tc* y, const Tv* vy, const T* xm,     \
                                                              const T* kx, const cstone::Box<Tc>& box,                 \
                                                              size_t startIndex, size_t endIndex)

GROWTH_GPU(double, double, double);
GROWTH_GPU(double, double, float);
GROWTH_GPU(double, float, float);
GROWTH_GPU(float, float, float);

//!@brief functor for the machSquare summing
template<class Tv, class T>
struct MachSquareSum
{
    HOST_DEVICE_FUN double operator()(const thrust::tuple<Tv, Tv, Tv, T>& p)
    {
        Vec3<Tv> V{get<0>(p), get<1>(p), get<2>(p)};
        auto     c = get<3>(p);
        return norm2(V) / (c * c);
    }
};

template<class Tv, class T>
double machSquareSumGpu(const Tv* vx, const Tv* vy, const Tv* vz, const T* c, size_t first, size_t last)
{
    auto it1 = thrust::make_zip_iterator(thrust::make_tuple(vx + first, vy + first, vz + first, c + first));
    auto it2 = thrust::make_zip_iterator(thrust::make_tuple(vx + last, vy + last, vz + last, c + last));

    auto   plus               = thrust::plus<double>{};
    double localMachSquareSum = 0.0;

    localMachSquareSum =
        thrust::transform_reduce(thrust::device, it1, it2, MachSquareSum<Tv, T>{}, localMachSquareSum, plus);

    return localMachSquareSum;
}

#define MACH_GPU(Tv, T)                                                                                                \
    template double machSquareSumGpu(const Tv* vx, const Tv* vy, const Tv* vz, const T* c, size_t, size_t)

MACH_GPU(double, double);
MACH_GPU(double, float);
MACH_GPU(float, float);

//!@brief functor to count particles that belong to the cloud
template<class T, class Tt, class Tm>
struct Survivors
{
    HOST_DEVICE_FUN
    double operator()(const thrust::tuple<Tt, T, T, Tm>& p)
    {
        size_t isCloud;
        auto   temp = get<0>(p);
        auto   kx   = get<1>(p);
        auto   xm   = get<2>(p);
        auto   m    = get<3>(p);
        auto   rhoi = kx / xm * m;
        if (rhoi >= 0.64 * rhoBubble && temp <= 0.9 * tempWind) { isCloud = 1; }
        else { isCloud = 0; }
        return isCloud;
    }

    double rhoBubble;
    double tempWind;
};

template<class T, class Tt, class Tm>
size_t survivorsGpu(const Tt* temp, const T* kx, const T* xmass, const Tm* m, double rhoBubble, double tempWind,
                    size_t first, size_t last)
{
    auto it1 = thrust::make_zip_iterator(thrust::make_tuple(temp + first, kx + first, xmass + first, m + first));
    auto it2 = thrust::make_zip_iterator(thrust::make_tuple(temp + last, kx + last, xmass + last, m + last));

    auto plus = thrust::plus<size_t>{};

    size_t localSurvivors = 0;

    localSurvivors = thrust::transform_reduce(thrust::device, it1, it2, Survivors<T, Tt, Tm>{rhoBubble, tempWind},
                                              localSurvivors, plus);

    return localSurvivors;
}

#define SURVIVORS(T, Tt, Tm)                                                                                           \
    template size_t survivorsGpu(const Tt* temp, const T* kx, const T* xmass, const Tm* m, double, double, size_t,     \
                                 size_t)

SURVIVORS(double, double, double);
SURVIVORS(float, float, float);
SURVIVORS(float, double, float);

template<class T, class Tc, class Tm>
struct MarkRampCond
{
    HOST_DEVICE_FUN
    void operator()(thrust::tuple<T, T, T, Tm, AuxT<T>&, AuxT<T>&> p)
    {
        T    h               = get<0>(p);
        T    y               = get<1>(p);
        T    vy              = get<2>(p);
        Tm   markRamp        = get<3>(p);
        bool closeToBoundary = thrust::min(std::abs(y - ymin), std::abs(y - ymax)) / h < 8.;
        if (markRamp > 0.05 && !closeToBoundary)
        {
            thrust::get<4>(p) = {y, vy};
            thrust::get<5>(p) = {y, vy};
        }
    }
    Tc ymin;
    Tc ymax;
};

template<class T, class Tc, class Th>
std::tuple<std::vector<AuxT<T>>, std::vector<AuxT<T>>>
localGrowthRateRTGpu(const int nAverage, size_t first, size_t last, Tc ymin, Tc ymax, const Th* h, const T* y,
                     const Th* vy, const Th* markRamp)
{
    thrust::device_vector<AuxT<T>> targetUp(last - first);
    thrust::device_vector<AuxT<T>> targetDown(last - first);

    auto it1 = thrust::make_zip_iterator(
        thrust::make_tuple(h + first, y + first, vy + first, markRamp + first, targetUp.begin(), targetDown.begin()));
    auto it2 = thrust::make_zip_iterator(
        thrust::make_tuple(h + last, y + last, vy + last, markRamp + last, targetUp.end(), targetDown.end()));

    thrust::for_each(thrust::device, it1, it2, MarkRampCond<T, Tc, Th>{ymin, ymax});

    auto endUp   = thrust::remove_if(thrust::device, targetUp.begin(), targetUp.end(), invalidAuxTEntry<T>());
    auto endDown = thrust::remove_if(thrust::device, targetDown.begin(), targetDown.end(), invalidAuxTEntry<T>());

    thrust::sort(thrust::device, targetUp.begin(), endUp, greaterRT());
    thrust::sort(thrust::device, targetDown.begin(), endDown, lowerRT());

    targetUp.resize(nAverage);
    targetDown.resize(nAverage);

    std::vector<AuxT<T>> retUp(nAverage);
    std::vector<AuxT<T>> retDown(nAverage);
    thrust::copy(targetUp.begin(), targetUp.end(), retUp.begin());
    thrust::copy(targetDown.begin(), targetDown.end(), retDown.begin());

    return std::make_tuple(retUp, retDown);
}

#define RTGROWTH(T, Tc, Th)                                                                                            \
    template std::tuple<std::vector<AuxT<T>>, std::vector<AuxT<T>>> localGrowthRateRTGpu(                              \
        const int nAverage, size_t first, size_t last, Tc ymin, Tc ymax, const Th* h, const T* y, const Th* vy,        \
        const Th* markRamp);

RTGROWTH(double, double, double);
RTGROWTH(double, float, double);
RTGROWTH(double, double, float);
RTGROWTH(double, float, float);
RTGROWTH(float, float, float);

} // namespace sphexa
